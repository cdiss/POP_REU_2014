#include "hip/hip_runtime.h"
#include <stdio.h>
#include "WKFUtils.h"

__global__ void kernelFloat1D(float startReal, float startImag, int steps, float step, unsigned* output, unsigned maxIters) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = idx / steps;
    int j = idx % steps;
    float real = startReal + step*j;
    float imag = startImag - step*i;
    unsigned iters = 0;
    float z_real = 0.0f, z_imag = 0.0f, z_sum = 0.0f;
    while (z_sum < 4.0f && iters < maxIters) {
        iters++;
        float z_real_sq = z_real*z_real;
        float z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;
    }
    output[idx] = iters;
}


__global__ void kernelFloat2D(float startReal, float startImag, float step, unsigned* output, unsigned maxIters) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = i * gridDim.x * blockDim.x + j;
    float real = startReal + step*j;
    float imag = startImag - step*i;
    unsigned iters = 0;
    float z_real = 0.0f, z_imag = 0.0f, z_sum = 0.0f;
    while (z_sum < 4.0f && iters < maxIters) {
        iters++;
        float z_real_sq = z_real*z_real;
        float z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;
    }
    output[idx] = iters;
}


__global__ void kernelFloat1D_unrolled(float startReal, float startImag, int steps, float step, unsigned* output, unsigned maxIters) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = idx / steps;
    int j = idx % steps;
    float real = startReal + step*j;
    float imag = startImag - step*i;
    unsigned iters = 0, iters_old = 0;
    float z_real = 0.0f, z_imag = 0.0f, z_sum = 0.0f;
    float z_real_old, z_imag_old, z_sum_old;

    while (z_sum < 4.0f && iters < maxIters) {
        iters_old = iters;
        z_real_old = z_real;
        z_imag_old = z_imag;
        z_sum_old = z_sum;        

        iters+=8;

        float z_real_sq = z_real*z_real;
        float z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;
    }

    iters = iters_old;
    z_real = z_real_old;
    z_imag = z_imag_old;
    z_sum = z_sum_old;
    while (z_sum < 4.0f && iters < maxIters) {
        iters++;
        float z_real_sq = z_real*z_real;
        float z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;
    }


    output[idx] = iters;
}

__global__ void kernelDouble2D_unrolled(double startReal, double startImag, double step, unsigned* output, unsigned maxIters) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = i * gridDim.x * blockDim.x + j;
    double real = startReal + step*j;
    double imag = startImag - step*i;
    unsigned iters = 0, iters_old = 0;
    double z_real = 0.0, z_imag = 0.0, z_sum = 0.0;
    double z_real_old, z_imag_old, z_sum_old;

    while (z_sum < 4.0f && iters < maxIters) {
        iters_old = iters;
        z_real_old = z_real;
        z_imag_old = z_imag;
        z_sum_old = z_sum;        

        iters+=8;

        double z_real_sq = z_real*z_real;
        double z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;

        z_real_sq = z_real*z_real;
        z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;
    }

    iters = iters_old;
    z_real = z_real_old;
    z_imag = z_imag_old;
    z_sum = z_sum_old;
    while (z_sum < 4.0f && iters < maxIters) {
        iters++;
        double z_real_sq = z_real*z_real;
        double z_imag_sq = z_imag*z_imag;
        z_sum = z_real_sq + z_imag_sq;
        z_imag = 2.0f*z_real*z_imag + imag;
        z_real = z_real_sq - z_imag_sq + real;
    }


    output[idx] = iters;
}


void cudaDouble(double startReal, double startImag, int steps, int horizsteps, double step, unsigned* output, unsigned maxiters) {

    hipError_t err;

    wkf_timerhandle timer = wkf_timer_create();
    
    int numPixels = steps*horizsteps;

    unsigned* d_output;
    err = hipMalloc(&d_output, (numPixels+511)*sizeof(unsigned));
    if (err != hipSuccess) {printf("Error with hipMalloc: %i\n", err); return;}
    
    wkf_timer_start(timer);
    dim3 dimGrid(horizsteps/16, steps/16, 1);
    dim3 dimBlock(16, 16, 1);
    kernelDouble2D_unrolled<<<dimGrid, dimBlock>>>(startReal, startImag, step, d_output, maxiters);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {printf("Error with hipMalloc: %i\n", err); return;}
    wkf_timer_stop(timer);
    printf("kernelDouble2D_unrolled: Time: %f\n", wkf_timer_time(timer));


    err = hipMemcpy(output, d_output, numPixels*sizeof(unsigned), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {printf("Error with hipMemcpy\n"); return;}

    err = hipFree(d_output);
    if (err != hipSuccess) {printf("Error with hipFree\n"); return;}

}

